/*
 * Copyright (c) 2020-2021 dePaul Miller (dsm220@lehigh.edu)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 */

#include <iostream>
#include <gtest/gtest.h>
#include "testheader.h"
#include <unordered_map>

using namespace lslab;

const int BLOCKS = 128;
const int THREADS_PER_BLOCK = 512;

TEST(slabunified_test, MemoryLeakageTest) {

    const int size = 1000;
    std::hash<unsigned> hfn;
    SlabUnified<unsigned, int *, BLOCKS, THREADS_PER_BLOCK> s(size);
    auto b = new BatchBuffer<unsigned, int *, BLOCKS, THREADS_PER_BLOCK>();

    s.setGPU();

    for (unsigned i = 0; i < (unsigned) size; i += THREADS_PER_BLOCK * BLOCKS) {
        unsigned j = 0;
        for (; j < THREADS_PER_BLOCK * BLOCKS && i * THREADS_PER_BLOCK * BLOCKS + j < size; j++) {
            unsigned key = 1;
            int *value = new int[256]; // allocating 1KB
            for (int w = 0; w < 256; w++) {
                value[w] = 1;
            }
            b->getBatchKeys()[j] = key;
            b->getHashValues()[j] = hfn(key);
            b->getBatchRequests()[j] = REQUEST_INSERT;
            b->getBatchValues()[j] = value;
        }
        for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
            b->getBatchRequests()[j] = REQUEST_EMPTY;
        }
        s.moveBufferToGPU(b, 0x0);
        s.diy_batch(b, BLOCKS, THREADS_PER_BLOCK, 0x0, 0);
        s.moveBufferToCPU(b, 0x0);
        gpuErrchk(hipStreamSynchronize(0x0));
        j = 0;
        for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
            if (b->getBatchRequests()[j] == REQUEST_INSERT && b->getBatchValues()[j] != nullptr) {
                delete[] b->getBatchValues()[j];
            }
        }
    }

    for (int rep = 0; rep < 100; rep++) {

        for (unsigned i = 0; i < (unsigned) size; i += THREADS_PER_BLOCK * BLOCKS) {
            unsigned j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS && i * THREADS_PER_BLOCK * BLOCKS + j < size; j++) {
                unsigned key = 1;
                int *value = new int[256]; // allocating 1KB
                for (int w = 0; w < 256; w++) {
                    value[w] = 1;
                }
                b->getBatchKeys()[j] = key;
                b->getHashValues()[j] = hfn(key);
                b->getBatchRequests()[j] = REQUEST_INSERT;
                b->getBatchValues()[j] = value;
            }
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                b->getBatchRequests()[j] = REQUEST_EMPTY;
            }
            s.moveBufferToGPU(b, 0x0);
            s.diy_batch(b, BLOCKS, THREADS_PER_BLOCK, 0x0, rep + 1);
            s.moveBufferToCPU(b, 0x0);
            gpuErrchk(hipStreamSynchronize(0x0));

            j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                if (b->getBatchRequests()[j] == REQUEST_INSERT && b->getBatchValues()[j] != nullptr) {
                    delete[] b->getBatchValues()[j];
                }
            }
        }
    }

    delete b;
}

TEST(slabunified_test, GetPutTest) {

    const int size = 1000;
    std::hash<unsigned> hfn;
    SlabUnified<unsigned, int *, BLOCKS, THREADS_PER_BLOCK> s(size);
    auto b = new BatchBuffer<unsigned, int *, BLOCKS, THREADS_PER_BLOCK>();

    s.setGPU();

    int count = 1;

    for (int rep = 0; rep < 100; rep++) {

        for (unsigned i = 0; i < (unsigned) size; i += THREADS_PER_BLOCK * BLOCKS) {
            unsigned j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS && i * THREADS_PER_BLOCK * BLOCKS + j < size; j++) {
                unsigned key = j;
                int *value = new int[256]; // allocating 1KB
                for (int w = 0; w < 256; w++) {
                    value[w] = rep;
                }
                b->getBatchKeys()[j] = key;
                b->getHashValues()[j] = hfn(key);
                b->getBatchRequests()[j] = REQUEST_INSERT;
                b->getBatchValues()[j] = value;
            }
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                b->getBatchRequests()[j] = REQUEST_EMPTY;
            }
            s.moveBufferToGPU(b, 0x0);
            s.diy_batch(b, BLOCKS, THREADS_PER_BLOCK, 0x0, count++);
            s.moveBufferToCPU(b, 0x0);
            gpuErrchk(hipStreamSynchronize(0x0));
            j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                if (b->getBatchRequests()[j] == REQUEST_INSERT && b->getBatchValues()[j] != nullptr) {

                    GTEST_ASSERT_NE(b->getBatchValues()[j], nullptr);
                    for (int w = 0; w < 256; w++) {
                        GTEST_ASSERT_EQ(b->getBatchValues()[j][w], rep - 1) << " old insert was rep - 1";
                    }

                    delete[] b->getBatchValues()[j];
                }
            }
        }

        for (unsigned i = 0; i < (unsigned) size; i += THREADS_PER_BLOCK * BLOCKS) {
            unsigned j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS && i * THREADS_PER_BLOCK * BLOCKS + j < size; j++) {
                unsigned key = j;
                b->getBatchKeys()[j] = key;
                b->getHashValues()[j] = hfn(key);
                b->getBatchRequests()[j] = REQUEST_GET;
            }
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                b->getBatchRequests()[j] = REQUEST_EMPTY;
            }
            s.moveBufferToGPU(b, 0x0);
            s.diy_batch(b, BLOCKS, THREADS_PER_BLOCK, 0x0, count++);
            s.moveBufferToCPU(b, 0x0);
            gpuErrchk(hipStreamSynchronize(0x0));

            j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                if (b->getBatchRequests()[j] == REQUEST_INSERT && b->getBatchValues()[j] != nullptr) {
                    delete[] b->getBatchValues()[j];
                }
                if (b->getBatchRequests()[j] == REQUEST_GET) {
                    GTEST_ASSERT_NE(b->getBatchValues()[j], nullptr);
                    for (int w = 0; w < 256; w++) {
                        GTEST_ASSERT_EQ(b->getBatchValues()[j][w], rep) << " last insert was rep";
                    }
                }
            }
        }
    }

    delete b;
}

TEST(slabunified_test, PutRemoveTest) {

    const int size = 1000;
    std::hash<unsigned> hfn;
    SlabUnified<unsigned, int *, BLOCKS, THREADS_PER_BLOCK> s(size);
    auto b = new BatchBuffer<unsigned, int *, BLOCKS, THREADS_PER_BLOCK>();

    s.setGPU();

    for (int rep = 0; rep < 100; rep++) {

        for (unsigned i = 0; i < (unsigned) size; i += THREADS_PER_BLOCK * BLOCKS) {
            unsigned j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS && i * THREADS_PER_BLOCK * BLOCKS + j < size; j++) {
                unsigned key = j + 1;
                int *value = new int[256]; // allocating 1KB
                for (int w = 0; w < 256; w++) {
                    value[w] = rep;
                }
                b->getBatchKeys()[j] = key;
                b->getHashValues()[j] = hfn(key);
                b->getBatchRequests()[j] = REQUEST_INSERT;
                b->getBatchValues()[j] = value;
            }
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                b->getBatchRequests()[j] = REQUEST_EMPTY;
            }
            s.moveBufferToGPU(b, 0x0);
            s.diy_batch(b, BLOCKS, THREADS_PER_BLOCK, 0x0, 2 * rep);
            s.moveBufferToCPU(b, 0x0);
            gpuErrchk(hipStreamSynchronize(0x0));
            j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                if (b->getBatchRequests()[j] == REQUEST_INSERT) {
                    GTEST_ASSERT_EQ(b->getBatchValues()[j], nullptr) << " should always be reading nullptr last; failure on key " << j + 1;
                }
            }
        }

        for (unsigned i = 0; i < (unsigned) size; i += THREADS_PER_BLOCK * BLOCKS) {
            unsigned j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS && i * THREADS_PER_BLOCK * BLOCKS + j < size; j++) {
                unsigned key = j + 1;
                b->getBatchKeys()[j] = key;
                b->getHashValues()[j] = hfn(key);
                b->getBatchRequests()[j] = REQUEST_REMOVE;
            }
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                b->getBatchRequests()[j] = REQUEST_EMPTY;
            }
            s.moveBufferToGPU(b, 0x0);
            s.diy_batch(b, BLOCKS, THREADS_PER_BLOCK, 0x0, 2 * rep + 1);
            s.moveBufferToCPU(b, 0x0);
            gpuErrchk(hipStreamSynchronize(0x0));

            j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                if (b->getBatchRequests()[j] == REQUEST_REMOVE) {
                    GTEST_ASSERT_NE(b->getBatchValues()[j], nullptr) << " key value pair was inserted on key " << j + 1;
                    for (int w = 0; w < 256; w++) {
                        GTEST_ASSERT_EQ(b->getBatchValues()[j][w], rep) << " last insert was rep";
                    }
                    delete[] b->getBatchValues()[j];
                }
            }
        }
    }

    delete b;
}

TEST(slabunified_test, PutRemoveTest_uint64) {


    const int size = 1000;
    std::hash<unsigned long long> hfn;
    SlabUnified<unsigned long long, int *, BLOCKS, THREADS_PER_BLOCK> s(size);
    auto b = new BatchBuffer<unsigned long long, int *, BLOCKS, THREADS_PER_BLOCK>();

    s.setGPU();

    for (int rep = 0; rep < 100; rep++) {

        for (unsigned i = 0; i < (unsigned) size; i += THREADS_PER_BLOCK * BLOCKS) {
            unsigned j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS && i * THREADS_PER_BLOCK * BLOCKS + j < size; j++) {
                unsigned long long key = j + 1;
                int *value = new int[256]; // allocating 1KB
                for (int w = 0; w < 256; w++) {
                    value[w] = rep;
                }
                b->getBatchKeys()[j] = key;
                b->getHashValues()[j] = hfn(key);
                b->getBatchRequests()[j] = REQUEST_INSERT;
                b->getBatchValues()[j] = value;
            }
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                b->getBatchRequests()[j] = REQUEST_EMPTY;
            }
            s.moveBufferToGPU(b, 0x0);
            s.diy_batch(b, BLOCKS, THREADS_PER_BLOCK, 0x0, 2 * rep);
            s.moveBufferToCPU(b, 0x0);
            gpuErrchk(hipStreamSynchronize(0x0));
            gpuErrchk(hipPeekAtLastError());
            j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                if (b->getBatchRequests()[j] == REQUEST_INSERT) {
                    GTEST_ASSERT_EQ(b->getBatchValues()[j], nullptr) << " should always be reading nullptr last";
                }
            }
        }

        for (unsigned i = 0; i < (unsigned) size; i += THREADS_PER_BLOCK * BLOCKS) {
            unsigned j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS && i * THREADS_PER_BLOCK * BLOCKS + j < size; j++) {
                unsigned long long key = j + 1;
                b->getBatchKeys()[j] = key;
                b->getHashValues()[j] = hfn(key);
                b->getBatchRequests()[j] = REQUEST_REMOVE;
            }
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                b->getBatchRequests()[j] = REQUEST_EMPTY;
            }
            s.moveBufferToGPU(b, 0x0);
            s.diy_batch(b, BLOCKS, THREADS_PER_BLOCK, 0x0, 2 * rep + 1);
            s.moveBufferToCPU(b, 0x0);
            gpuErrchk(hipStreamSynchronize(0x0));

            j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                if (b->getBatchRequests()[j] == REQUEST_REMOVE) {
                    GTEST_ASSERT_NE(b->getBatchValues()[j], nullptr) << " key value pair was inserted on key";
                    for (int w = 0; w < 256; w++) {
                        GTEST_ASSERT_EQ(b->getBatchValues()[j][w], rep) << " last insert was rep";
                    }
                    delete[] b->getBatchValues()[j];
                }
            }
        }
    }

    delete b;
}

LSLAB_HOST_DEVICE int memcmp_(const volatile void* a, const volatile void* b, size_t size) noexcept {
    for(size_t i = 0; i < size; i++) {
        char diff = reinterpret_cast<const volatile char*>(a)[i] - reinterpret_cast<const volatile char*>(b)[i];
        if(diff != 0) return diff;
    }
    return 0;
}

struct Key {

    LSLAB_HOST_DEVICE constexpr Key() : bytes{0} {}

    LSLAB_HOST_DEVICE Key(const Key& k) {
        memcpy(bytes, k.bytes, sizeof(bytes));
    }

    LSLAB_HOST_DEVICE Key(const volatile Key& k) {
        for(int i = 0; i < 128; i++) {
            bytes[i] = k.bytes[i];
        }
    }


    LSLAB_HOST_DEVICE Key(unsigned long long i) {
        memset(bytes, 0, sizeof(bytes));
        memcpy(bytes, &i, sizeof(unsigned long long));
    }

    LSLAB_HOST_DEVICE bool operator==(const Key k) const volatile {
        return memcmp_(bytes, k.bytes, 128) == 0;
    }

    LSLAB_HOST_DEVICE volatile Key& operator=(const Key& k) volatile {
        for(int i = 0; i < 128; i++) {
            bytes[i] = k.bytes[i];
        }
        return *this;
    }


    LSLAB_HOST_DEVICE volatile Key& operator=(const volatile Key& k) volatile {
        for(int i = 0; i < 128; i++) {
            bytes[i] = k.bytes[i];
        }
        return *this;
    }

    friend std::ostream& operator<<(std::ostream&, const Key&);

    alignas(128) char bytes[128];
};

std::ostream& operator<<(std::ostream& s, const Key& k) {
    unsigned long long i = 0;
    memcpy(&i, k.bytes, sizeof(unsigned long long));
    s << i;
    return s;
}

namespace std {

template<>
struct hash<Key> {

    std::size_t operator()(const Key& k) const {
        return k.bytes[0];
    }

};

}

TEST(slabunified_test, PutRemoveTest_128B) {


    const int BLOCKS_ = 128;
    const int CHOSEN_THREADS_PER_BLOCK = 32;

    const int size = 1000;
    std::hash<Key> hfn;
    SlabUnified<Key, int *, BLOCKS_, CHOSEN_THREADS_PER_BLOCK> s(size);
    auto b = new BatchBuffer<Key, int *, BLOCKS_, CHOSEN_THREADS_PER_BLOCK>();

    s.setGPU();

    std::unordered_map<Key, int*> reference;
    std::unordered_map<int*, Key> reverse;

    for (int rep = 0; rep < 100; rep++) {

        for (unsigned i = 0; i < (unsigned) size; i += CHOSEN_THREADS_PER_BLOCK * BLOCKS_) {
            unsigned j = 0;
            for (; j < CHOSEN_THREADS_PER_BLOCK * BLOCKS_ && i * CHOSEN_THREADS_PER_BLOCK * BLOCKS_ + j < size; j++) {
                unsigned long long key = j + 1;
                int *value = new int[256]; // allocating 1KB
                reference[key] = value;
                reverse[value] = key;
                //std::cerr << "(" << key << "," << (void*)value << ")" << std::endl;
                for (int w = 0; w < 256; w++) {
                    value[w] = rep;
                }
                b->getBatchKeys()[j] = key;
                b->getHashValues()[j] = hfn(key);
                b->getBatchRequests()[j] = REQUEST_INSERT;
                b->getBatchValues()[j] = value;
                value = nullptr;
            }
            for (; j < CHOSEN_THREADS_PER_BLOCK * BLOCKS_; j++) {
                b->getBatchRequests()[j] = REQUEST_EMPTY;
                b->getBatchValues()[j] = nullptr;
            }
            s.moveBufferToGPU(b, 0x0);
            s.diy_batch(b, BLOCKS_, CHOSEN_THREADS_PER_BLOCK, 0x0, 2 * rep);
            s.moveBufferToCPU(b, 0x0);
            gpuErrchk(hipStreamSynchronize(0x0));
            gpuErrchk(hipPeekAtLastError());
            j = 0;
            for (; j < CHOSEN_THREADS_PER_BLOCK * BLOCKS_; j++) {
                if (b->getBatchRequests()[j] == REQUEST_INSERT) {
                    GTEST_ASSERT_EQ(b->getBatchValues()[j], nullptr) << " should always be reading nullptr last. Found incorrect at " << j;
                }
            }
        }

        for (unsigned i = 0; i < (unsigned) size; i += CHOSEN_THREADS_PER_BLOCK * BLOCKS_) {
            unsigned j = 0;
            for (; j < CHOSEN_THREADS_PER_BLOCK * BLOCKS_ && i * CHOSEN_THREADS_PER_BLOCK * BLOCKS_ + j < size; j++) {
                unsigned key = j + 1;
                b->getBatchKeys()[j] = key;
                b->getHashValues()[j] = hfn(key);
                b->getBatchRequests()[j] = REQUEST_REMOVE;
                b->getBatchValues()[j] = nullptr; // to catch errors
            }
            for (; j < CHOSEN_THREADS_PER_BLOCK * BLOCKS_; j++) {
                b->getBatchRequests()[j] = REQUEST_EMPTY;
                b->getBatchValues()[j] = nullptr; // to catch errors
            }
            s.moveBufferToGPU(b, 0x0);
            s.diy_batch(b, BLOCKS_, CHOSEN_THREADS_PER_BLOCK, 0x0, 2 * rep + 1);
            s.moveBufferToCPU(b, 0x0);
            gpuErrchk(hipStreamSynchronize(0x0));
            gpuErrchk(hipPeekAtLastError());

            j = 0;
            for (; j < CHOSEN_THREADS_PER_BLOCK * BLOCKS_; j++) {
                if (b->getBatchRequests()[j] == REQUEST_REMOVE) {
                    GTEST_ASSERT_NE(b->getBatchValues()[j], nullptr) << " when removing found nullptr at " << j << " for reference it is " << (void*)reference[j];
                    GTEST_ASSERT_EQ(b->getBatchValues()[j], reference[b->getBatchKeys()[j]]) << " batch values should equal reference at j = " << j << " but found " << reverse[b->getBatchValues()[j]]; 
                    for (int w = 0; w < 256; w++) {
                        GTEST_ASSERT_EQ(b->getBatchValues()[j][w], rep) << " last insert was " << rep << " pointer is " << b->getBatchValues()[j] << " j is " << j << " w is " << w << " the pointer should be " << (void*) reference[j] << " and it is" << (b->getBatchValues()[j] == reference[b->getBatchKeys()[j]] ? "" : " not");
                    }
                    delete[] b->getBatchValues()[j];
                    b->getBatchValues()[j] = nullptr;
                }
            }
        }
    }

    delete b;
}

TEST(slabunified_test, PutRemoveTest_128Bto128B) {


    const int BLOCKS_ = 10;
    const int CHOSEN_THREADS_PER_BLOCK = 32;

    const int size = 1000;
    std::hash<Key> hfn;
    SlabUnified<Key, Key, BLOCKS_, CHOSEN_THREADS_PER_BLOCK> s(size);
    auto b = new BatchBuffer<Key, Key, BLOCKS_, CHOSEN_THREADS_PER_BLOCK>();

    s.setGPU();

    std::unordered_map<Key, Key> reference;
    std::unordered_map<Key, Key> reverse;

    for (int rep = 0; rep < 100; rep++) {

        for (unsigned i = 0; i < (unsigned) size; i += CHOSEN_THREADS_PER_BLOCK * BLOCKS_) {
            unsigned j = 0;
            for (; j < CHOSEN_THREADS_PER_BLOCK * BLOCKS_ && i * CHOSEN_THREADS_PER_BLOCK * BLOCKS_ + j < size; j++) {
                unsigned long long key = j + 1;
                unsigned long long value = rep;
                reference[key] = value;
                reverse[value] = key;
                //std::cerr << "(" << key << "," << (void*)value << ")" << std::endl;
                b->getBatchKeys()[j] = key;
                b->getHashValues()[j] = hfn(key);
                b->getBatchRequests()[j] = REQUEST_INSERT;
                b->getBatchValues()[j] = value;
            }
            for (; j < CHOSEN_THREADS_PER_BLOCK * BLOCKS_; j++) {
                b->getBatchRequests()[j] = REQUEST_EMPTY;
            }
            s.moveBufferToGPU(b, 0x0);
            s.diy_batch(b, BLOCKS_, CHOSEN_THREADS_PER_BLOCK, 0x0, 2 * rep);
            s.moveBufferToCPU(b, 0x0);
            gpuErrchk(hipStreamSynchronize(0x0));
            gpuErrchk(hipPeekAtLastError());
            j = 0;
        }

        for (unsigned i = 0; i < (unsigned) size; i += CHOSEN_THREADS_PER_BLOCK * BLOCKS_) {
            unsigned j = 0;
            for (; j < CHOSEN_THREADS_PER_BLOCK * BLOCKS_ && i * CHOSEN_THREADS_PER_BLOCK * BLOCKS_ + j < size; j++) {
                unsigned key = j + 1;
                b->getBatchKeys()[j] = key;
                b->getHashValues()[j] = hfn(key);
                b->getBatchRequests()[j] = REQUEST_REMOVE;
                b->getBatchValues()[j] = 0; // to catch errors
            }
            for (; j < CHOSEN_THREADS_PER_BLOCK * BLOCKS_; j++) {
                b->getBatchRequests()[j] = REQUEST_EMPTY;
                b->getBatchValues()[j] = 0; // to catch errors
            }
            s.moveBufferToGPU(b, 0x0);
            s.diy_batch(b, BLOCKS_, CHOSEN_THREADS_PER_BLOCK, 0x0, 2 * rep + 1);
            s.moveBufferToCPU(b, 0x0);
            gpuErrchk(hipStreamSynchronize(0x0));
            gpuErrchk(hipPeekAtLastError());

            j = 0;
        }
    }

    delete b;
}

TEST(slabunified_test, GetAllPutTest) {

    const int size = 50;
    std::hash<unsigned> hfn;
    SlabUnified<unsigned, int *, BLOCKS, THREADS_PER_BLOCK> s(size);
    auto b = new BatchBuffer<unsigned, int *, BLOCKS, THREADS_PER_BLOCK>();

    s.setGPU();

    for (int rep = 0; rep < 100; rep++) {

        for (unsigned i = 0; i < (unsigned) size; i += THREADS_PER_BLOCK * BLOCKS) {
            unsigned j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS && i * THREADS_PER_BLOCK * BLOCKS + j < size; j++) {
                unsigned key = j;
                int *value = new int[256]; // allocating 1KB
                for (int w = 0; w < 256; w++) {
                    value[w] = rep;
                }
                std::cerr << "INSERT " << key << " " << (void*) value << std::endl;
                b->getBatchKeys()[j] = key;
                b->getHashValues()[j] = hfn(key);
                b->getBatchRequests()[j] = REQUEST_INSERT;
                b->getBatchValues()[j] = value;
            }
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                b->getBatchRequests()[j] = REQUEST_EMPTY;
            }
            s.moveBufferToGPU(b, 0x0);
            s.diy_batch(b, BLOCKS, THREADS_PER_BLOCK, 0x0, 2 * rep + 1);
            s.moveBufferToCPU(b, 0x0);
            gpuErrchk(hipStreamSynchronize(0x0));
            j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                if (b->getBatchRequests()[j] == REQUEST_INSERT && b->getBatchValues()[j] != nullptr) {

                    GTEST_ASSERT_NE(b->getBatchValues()[j], nullptr);
                    for (int w = 0; w < 256; w++) {
                        GTEST_ASSERT_EQ(b->getBatchValues()[j][w], rep - 1) << " old insert was rep - 1";
                    }

                    delete[] b->getBatchValues()[j];
                }
            }
        }

        unsigned* allKeys;
        int** allValues;

        size_t size = s.getTable(allKeys, allValues, BLOCKS, THREADS_PER_BLOCK, 0x0);

        for(size_t i = 0; i < size; i++) {
            std::cerr << "GOT " << allKeys[i] << " " << (void*) allValues[i] << std::endl;
            for(int j = 0; j < 256; j++) {
                GTEST_ASSERT_EQ(allValues[i][j], rep) << " last insert was rep";
            }
        }

        delete[] allKeys;
        delete[] allValues;
    }

    delete b;
}

TEST(slabunified_test, PutRemoveTest_128B_Heavy) {


    const int BLOCKS_ = 1;
    const int CHOSEN_THREADS_PER_BLOCK = 64;

    const int size = 1;
    std::hash<Key> hfn;
    SlabUnified<Key, int *, BLOCKS_, CHOSEN_THREADS_PER_BLOCK> s(size);
    auto b = new BatchBuffer<Key, int *, BLOCKS_, CHOSEN_THREADS_PER_BLOCK>();

    s.setGPU();

    std::unordered_map<Key, int*> reference;
    std::unordered_map<int*, Key> reverse;

    for (int rep = 0; rep < 10; rep++) {

        for (unsigned i = 0; i < CHOSEN_THREADS_PER_BLOCK * BLOCKS_; i++) {
            unsigned long long key = i + 1 + rep * CHOSEN_THREADS_PER_BLOCK * BLOCKS_;
            int *value = new int[256]; // allocating 1KB
            reference[key] = value;
            reverse[value] = key;
            //std::cerr << "(" << key << "," << (void*)value << ")" << std::endl;
            for (int w = 0; w < 256; w++) {
                value[w] = rep;
            }
            b->getBatchKeys()[i] = key;
            b->getHashValues()[i] = hfn(key);
            b->getBatchRequests()[i] = REQUEST_INSERT;
            b->getBatchValues()[i] = value;
            value = nullptr;
        }
        s.moveBufferToGPU(b, 0x0);
        s.diy_batch(b, BLOCKS_, CHOSEN_THREADS_PER_BLOCK, 0x0, 2 * rep);
        s.moveBufferToCPU(b, 0x0);
        gpuErrchk(hipStreamSynchronize(0x0));
        gpuErrchk(hipPeekAtLastError());
        for (int j = 0; j < CHOSEN_THREADS_PER_BLOCK * BLOCKS_; j++) {
            delete[] b->getBatchValues()[j];
        }
    }

    delete b;
}

