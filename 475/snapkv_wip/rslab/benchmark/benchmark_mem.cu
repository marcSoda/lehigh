//
// Created by depaulsmiller on 9/3/20.
//

#include <LSlab/StandardSlabDefinitions.h>
#include <vector>
#include <LSlab/Slab.h>
#include <hip/hip_runtime_api.h>
#include <unordered_map>

using namespace lslab;

const int BLOCKS = 128;
const int THREADS_PER_BLOCK = 512;

int main() {

    const int size = 1000;
    std::hash<unsigned> hfn;
    SlabUnified<unsigned, int *, BLOCKS, THREADS_PER_BLOCK> s(size);
    auto b = new BatchBuffer<unsigned, int *, BLOCKS, THREADS_PER_BLOCK>();

    s.setGPU();

    for (unsigned i = 0; i < (unsigned) size; i += THREADS_PER_BLOCK * BLOCKS) {
        unsigned j = 0;
        for (; j < THREADS_PER_BLOCK * BLOCKS && i * THREADS_PER_BLOCK * BLOCKS + j < size; j++) {
            unsigned key = 1;
            int *value = new int[256]; // allocating 1KB
            for (int w = 0; w < 256; w++) {
                value[w] = 1;
            }
            b->getBatchKeys()[j] = key;
            b->getHashValues()[j] = hfn(key);
            b->getBatchRequests()[j] = REQUEST_INSERT;
            b->getBatchValues()[j] = value;
        }
        for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
            b->getBatchRequests()[j] = REQUEST_EMPTY;
        }
        s.moveBufferToGPU(b, 0x0);
        s.diy_batch(b, BLOCKS, THREADS_PER_BLOCK, 0x0, 1);
        s.moveBufferToCPU(b, 0x0);
        gpuErrchk(hipStreamSynchronize(0x0));
        j = 0;
        for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
            if (b->getBatchRequests()[j] == REQUEST_INSERT && b->getBatchValues()[j] != nullptr) {
                delete[] b->getBatchValues()[j];
            }
        }
    }

    gpuErrchk(hipProfilerStart());

    for (int rep = 0; rep < 10; rep++) {

        for (unsigned i = 0; i < (unsigned) size; i += THREADS_PER_BLOCK * BLOCKS) {
            unsigned j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS && i * THREADS_PER_BLOCK * BLOCKS + j < size; j++) {
                unsigned key = 1;
                int *value = new int[256]; // allocating 1KB
                for (int w = 0; w < 256; w++) {
                    value[w] = 1;
                }
                b->getBatchKeys()[j] = key;
                b->getHashValues()[j] = hfn(key);
                b->getBatchRequests()[j] = REQUEST_INSERT;
                b->getBatchValues()[j] = value;
            }
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                b->getBatchRequests()[j] = REQUEST_EMPTY;
            }
            s.moveBufferToGPU(b, 0x0);
            s.diy_batch(b, BLOCKS, THREADS_PER_BLOCK, 0x0, 1);
            s.moveBufferToCPU(b, 0x0);
            gpuErrchk(hipStreamSynchronize(0x0));

            j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                if (b->getBatchRequests()[j] == REQUEST_INSERT && b->getBatchValues()[j] != nullptr) {
                    delete[] b->getBatchValues()[j];
                }
            }
        }
    }

    gpuErrchk(hipProfilerStop());
    delete b;
}
